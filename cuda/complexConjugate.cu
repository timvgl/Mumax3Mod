
#include <hip/hip_runtime.h>
extern "C" __global__ void
complexConjugate(float* __restrict__ output, float* __restrict__ input, int N) {
    int idx = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (idx < N) {
        // Store amplitude and phase alternately
        output[2 * idx] = input[2 * idx];
        output[2 * idx + 1] = -input[2 * idx + 1];
    }
}