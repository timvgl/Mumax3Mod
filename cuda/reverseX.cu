#include "hip/hip_runtime.h"
// fftshift for 3D data, shifting only y and z axes
#include "stencil.h" 
extern "C" __global__ void
reverseX(float* __restrict__ data_out, float* __restrict__ data_in,
                                   int Nx, int Ny, int Nz, int oddNx) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // x index (first axis)
    int y = blockIdx.y * blockDim.y + threadIdx.y; // y index (second axis)
    int z = blockIdx.z * blockDim.z + threadIdx.z; // z index (third axis)
    int maxNx = oddNx == 1 ? Nx / 2 +1 : Nx / 2;
    if (x < maxNx && y < Ny && z < Nz) {
        // First axis (x) is not shifted
        int x_left = 2*x;
        int x_right = 2*Nx - 2*x - 2;
        int mid = oddNx == 1 ? Nx : -1;

        // Calculate linear indices for input and output arrays
        int idx_leftReal = index(x_left, y, z, 2*Nx, Ny, Nz);
        int idx_leftImag = index(x_left+1, y, z, 2*Nx, Ny, Nz);
        int idx_rightReal = index(x_right, y, z, 2*Nx, Ny, Nz);
        int idx_rightImag = index(x_right+1, y, z, 2*Nx, Ny, Nz);
        float dataLeftReal = data_in[idx_leftReal];
        float dataLeftImag = data_in[idx_leftImag];
        float dataRightReal = data_in[idx_rightReal];
        float dataRightImag = data_in[idx_rightImag];
        if (x_left == mid) {
            data_out[idx_leftReal] = dataLeftReal;
            data_out[idx_leftImag] = dataLeftImag;
            data_out[idx_rightReal] = dataRightReal;
            data_out[idx_rightImag] = dataRightImag;
        } else {
            // Copy data from input to output at the shifted position
            data_out[idx_leftReal] = dataRightReal;
            data_out[idx_leftImag] = dataRightImag;
            data_out[idx_rightReal] = dataLeftReal;
            data_out[idx_rightImag] = dataLeftImag;
        }
    }
}
//NxTotal = 256
//Nx = 64
//x_left = 2*x
//x_right = 4*(Nx - x/2) -2
//x = 0
//x_left = 0
//x_right = 4*(64 - 0) -2 = 254
//x = 1
//x_left = 2
//x_right = 4*(64 - 1/2) -2 = 252
//x = 2
//x_left = 4
//x_right = 4*(64 - 1) -2 = 250