#include <hip/hip_runtime.h>
extern "C" __global__ void
YnGovaluate1X3(float* __restrict__ output, float input2, float* __restrict__ input, int N) {
    int idx = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if (idx < N) {
        output[idx] = ynf(int(input2), input[idx]);
    }
}