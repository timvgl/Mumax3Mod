
#include <hip/hip_runtime.h>
extern "C" __global__ void erfGovaluate(float* __restrict__ value, int N) {
    int idx = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if (idx < N) {
        value[idx] = erff(value[idx]);
    }
}
