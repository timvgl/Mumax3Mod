#include "hip/hip_runtime.h"
#include "stencil.h"
#include <hip/hip_complex.h>

// Berechnet exp(z) für komplexe z = z.x + i*z.y.
__device__ __forceinline__ hipDoubleComplex my_cexp(hipDoubleComplex z) {
    hipDoubleComplex res;
    double t = expf(z.x);
    sincos(z.y, &res.y, &res.x); // berechnet cos(z.y) -> res.x und sin(z.y) -> res.y
    res.x *= t;
    res.y *= t;
    return res;
}

// Kernel zur inkrementellen NUDFT-Update.
// Parameter:
//  - sum: 4D-Array (Raum: (Nx/2, Ny, Nz), Frequenz: Nf) – gespeicherte Summe, komplex (interleaved: Real, Imag)
//  - src: 3D-Array mit neuen Daten, komplex (interleaved), Dimensionen: (Nx/2, Ny, Nz)
//  - Nx: Länge der x-Achse in Floats (also 2*(Anzahl komplexer Elemente in x))
//  - Ny, Nz: Anzahl der Elemente in y und z (in komplexen Elementen)
//  - Nf: Anzahl der Frequenzindizes, für die die NUDFT evaluiert wird
//  - minF, dF: Parameter zur Berechnung der Frequenz (f = minF + dF * fi)
//  - t: Zeitpunkt bzw. Zeitparameter des aktuellen Datenblocks
extern "C" __global__ void
FFT_Step_MEM_Complex(
    float* __restrict__ sum,   // 4D Summe (in interleaved Complex)
    float* __restrict__ src,   // 3D neue Daten (in interleaved Complex)
    int Nx, int Ny, int Nz,     // Nx: Länge in Floats, also x enthält Nx/2 komplexe Werte
    int Nf,
    float minF, float dF, float t)
{
    // Berechne die Anzahl komplexer Elemente in x.
    const double pi = 3.14159265358979323846;
    int Nx_c = Nx / 2;
    
    // Berechne die 3D-Raumkoordinaten (für das 3D-Array src)
    int x = blockIdx.x * blockDim.x + threadIdx.x;  // x als Index in komplexen Elementen
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (x >= Nx_c || y >= Ny || z >= Nz)
        return;
    
    // Berechne den linearen Index für src (3D-Array) – interleaved komplex:
    // Hier liefert das Makro idx(2*x, y, z) den Index für den Realteil,
    // da Nx in idx() die Länge in Floats ist (also 2*Nx_c).
    int srcIndex = idx(2 * x, y, z);  // Realteil an srcIndex, Imag an srcIndex+1
    hipDoubleComplex newData = make_hipDoubleComplex((double)src[srcIndex], (double)src[srcIndex + 1]);
    // Aktualisiere für jeden Frequenzindex die Summe:
    for (int fi = 0; fi < Nf; fi++) {
        // Berechne den Phasenwinkel für diese Frequenz:
        double phase = -2.0 * pi * ((double)minF + (double)dF * (double)fi) * (double)t;
        hipDoubleComplex expVal = my_cexp(make_hipDoubleComplex(0.0, phase));
        
        // Komplexe Multiplikation: Beitrag = newData * expVal
        hipDoubleComplex contribution;
        contribution.x = newData.x * expVal.x - newData.y * expVal.y;
        contribution.y = newData.x * expVal.y + newData.y * expVal.x;
        
        // Berechne den 4D-Index für die Summe:
        // Das 4D-Array hat Dimensionen: (Nx/2, Ny, Nz, Nf) in komplexen Elementen,
        // wobei die erste Dimension (x) in Floats als 2*(Nx/2) gespeichert wird.
        // Wir verwenden idx4D, wobei wir als x-Wert wieder 2*x verwenden, um den Realteil zu adressieren.
        int sumIndex = idx4D(2 * x, y, z, fi);  // Realteil
        // Akkumulieren: addiere den Beitrag zur vorhandenen Summe.
        sum[sumIndex]     += (float)contribution.x;
        sum[sumIndex + 1] += (float)contribution.y;
    }
}
