
#include <hip/hip_runtime.h>
extern "C" __global__ void atanhGovaluate(float* __restrict__ value, int N) {
    int idx = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if (idx < N) {
        value[idx] = atanhf(value[idx]);
    }
}
