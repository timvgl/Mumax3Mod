
#include <hip/hip_runtime.h>
extern "C" __global__ void
divGovaluate1X3(float* __restrict__ output, float input2, float* __restrict__ input, int N) {
    int idx = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if (idx < N) {
        output[idx] = input2 / input[idx];
    }
}