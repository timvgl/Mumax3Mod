
#include <hip/hip_runtime.h>
extern "C" __global__ void ceilGovaluate(float* __restrict__ value, int N) {
    int idx = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if (idx < N) {
        value[idx] = ceilf(value[idx]);
    }
}
