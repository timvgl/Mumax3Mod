#include "hip/hip_runtime.h"
#include "stencil.h"
#include <hip/hip_complex.h>

__device__ __forceinline__ hipComplex my_cexpf (hipComplex z) {
    hipComplex res;
    float t = expf (z.x);
    sincosf (z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

extern "C" __global__ void
FFT_Step_Real(float* __restrict__ dst, float* __restrict__ src1, float* __restrict__ src2, int Nx, int Ny, int Nz, float phase, float n) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }
    int IReal = idx(2*ix, iy, iz);
    int IImag = idx(2*ix+1, iy, iz);
    hipComplex src2Complex = make_hipComplex(src2[idx(ix, iy, iz)], 0.);
    hipComplex angle = make_hipComplex(0, phase);
    hipComplex expComplex = my_cexpf(angle);
    float dstReal = src2Complex.x * expComplex.x - src2Complex.y * expComplex.y; // Real part
    float dstImag = src2Complex.x * expComplex.y + src2Complex.y * expComplex.x; // Imaginary part
    dst[IReal] = dstReal / n + src1[IReal];
    dst[IImag] = dstImag / n + src1[IImag];
}
