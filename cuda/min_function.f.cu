#include "hip/hip_runtime.h"
#include "reduce.h"
#include "atomicf.h"
#include "float3.h"
#include "min.h"

#define load(i)  \
	        a[i]

extern "C" __global__ void minGovaluate(float* __restrict__ a,
    float* __restrict__ dst, float initVal, int n) {
        reduce(load, mymin, atomicFmin)
}
