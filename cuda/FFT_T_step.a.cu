#include "hip/hip_runtime.h"
#include "stencil.h"
#include <hip/hip_complex.h>
#include <stdint.h>

__device__ __forceinline__ hipComplex my_cexpf (hipComplex z) {
    hipComplex res;
    float t = expf (z.x);
    sincosf (z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

__device__ __forceinline__ hipComplex FFT_Stepper (float angleReal, float angleImag, hipComplex src2Complex) {
    hipComplex angle = make_hipComplex(angleReal, angleImag);
    hipComplex expComplex = my_cexpf(angle);
    float dstReal = src2Complex.x * expComplex.x - src2Complex.y * expComplex.y; // Real part
    float dstImag = src2Complex.x * expComplex.y + src2Complex.y * expComplex.x; // Imaginary part
    return make_hipComplex(dstReal, dstImag);
}


extern "C" __global__ void
FFT_Step(uint8_t* __restrict__ dst, uint8_t* __restrict__ src1, uint8_t* __restrict__ src2, int Nx, int Ny, int Nz, float angleReal, float angleImag, int n) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx / 2 || iy >= Ny || iz >= Nz) {
        return;
    }
    int IReal = idx(2*ix, iy, iz);
    int IImag = idx(2*ix+1, iy, iz);
    hipComplex src2Complex = make_hipComplex(src2[IReal], src2[IImag]);
    hipComplex dstVal = FFT_Stepper(angleReal, angleImag, src2Complex);
    
    dst[IReal] = dstVal.x / n + src1[IReal];
    dst[IImag] = dstVal.y / n + src1[IImag];
}