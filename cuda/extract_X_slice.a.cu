
#include <hip/hip_runtime.h>
extern "C" __global__ void
extractXSlice(
    float* __restrict__ output,       // Flattened 3D input array
    float* __restrict__ input,        // Output array for the X slice
    int X,                            // Size along the X dimension
    int Y,                            // Size along the Y dimension
    int Z,                            // Size along the Z dimension
    int y,                            // Fixed Y index
    int z                             // Fixed Z index
) {
    // Calculate the thread's X index
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't go out of bounds
    if (x < X) {
        // Compute the 1D index for the (x, y, z) element
        int index = x * (Y * Z) + y * Z + z;

        // Retrieve the value from the input and store it in the output
        output[x] = input[index];
    }
}

extern "C" __global__ void
extractYSlice(
    float* __restrict__ output,       // Output array for the Y slice
    float* __restrict__ input,        // Flattened 3D input array
    int X,                            // Size along the X dimension
    int Y,                            // Size along the Y dimension
    int Z,                            // Size along the Z dimension
    int x,                            // Fixed X index
    int z                             // Fixed Z index
) {
    // Calculate the thread's Y index
    int y = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't go out of bounds
    if (y < Y) {
        // Compute the 1D index for the (x, y, z) element
        int index = x * (Y * Z) + y * Z + z;

        // Retrieve the value from the input and store it in the output
        output[y] = input[index];
    }
}

extern "C" __global__ void
extractZSlice(
    float* __restrict__ output,      // Output array for the Z slice
    float* __restrict__ input,       // Flattened 3D input array
    int X,                           // Size along the X dimension
    int Y,                           // Size along the Y dimension
    int Z,                           // Size along the Z dimension
    int x,                           // Fixed X index
    int y                            // Fixed Y index
) {
    // Calculate the thread's Z index
    int z = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't go out of bounds
    if (z < Z) {
        // Compute the 1D index for the (x, y, z) element
        int index = x * (Y * Z) + y * Z + z;

        // Retrieve the value from the input and store it in the output
        output[z] = input[index];
    }
}